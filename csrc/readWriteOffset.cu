#include "hip/hip_runtime.h"
#include "../include/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

void initialData(float *A, const unsigned  long int N)
{
    for (unsigned long int i = 0; i < N; i++)
    {
        A[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

__global__ void readOffset(float *A, float *B, float *C, const unsigned  long int N, const int offset)
{
    unsigned long int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long int k = tid + offset;
    if (k < N)
        C[tid] = A[k] + B[k];
}

int main(int argc, char *argv[])
{
    // set up the device
    int dev = 0;
    hipDeviceProp_t devProp;

    CHECK(hipSetDevice(dev));
    CHECK(cudaGetDeviceProperties_v2(&devProp, dev));
    printf("Using device %d : %s\n", dev, devProp.name);

    // set up array size
    unsigned long int nElems = 1 << 24;
    size_t nBytes = nElems * sizeof(float);
    printf("Array size %ld of %zu bytes\n", nElems, nBytes);

    // set block size and offset 
    // expect offset format 0,11,128
    char *input = argv[1];
    int blockSize = atoi(argv[2]);

    // execution configuration
    dim3 block(blockSize);
    dim3 grid((nElems + block.x - 1) / block.x);

    // allocate the host memory
    float *h_A, *h_B;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);

    // initial data
    initialData(h_A, nElems);
    initialData(h_B, nElems);

    // allocate the device memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void **)&d_A, nBytes));
    CHECK(hipMalloc((void **)&d_B, nBytes));
    CHECK(hipMalloc((void **)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    char *token;
    int offset;
    token = strtok(input, ",");
    while (token != NULL)
    {   
        offset = atoi(token);
        // launch the kernel
        printf("Launch Kernel<<<%d,%d>>> with %d offset\n",grid.x,block.x,offset);
        readOffset<<<grid, block>>>(d_A, d_B, d_C, nElems, offset);
        CHECK(hipDeviceSynchronize());

        token = strtok(NULL,",");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}