#include "hip/hip_runtime.h"
#include "../include/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

void initialData(float *A, const unsigned long int N)
{
    for (unsigned long int i = 0; i < N; i++)
    {
        A[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

__global__ void readOffset(float *A, float *B, float *C, const unsigned long int N, const int offset)
{
    unsigned long int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long int k = tid + offset;
    if (k < N)
        C[tid] = A[k] + B[k];
}

__global__ void readOnlyCache(float *A, float *B, float *C, const unsigned long int N, const int offset)
{
    unsigned long tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long int k = tid + offset;
    if (k < N)
        C[tid] = __ldg(&A[k]) + __ldg(&B[k]);
}

int main(int argc, char *argv[])
{
    // set up the device
    int dev = 0;
    hipDeviceProp_t devProp;

    CHECK_ERROR(hipSetDevice(dev));
    CHECK_ERROR(cudaGetDeviceProperties_v2(&devProp, dev));
    printf("Using device %d : %s\n", dev, devProp.name);

    // set up array size
    unsigned long int nElems = 1 << 24;
    size_t nBytes = nElems * sizeof(float);
    printf("Array size %ld of %zu bytes\n", nElems, nBytes);

    // set block size and offset
    // expect offset format 0,11,128
    char *input = argv[1];
    int blockSize = atoi(argv[2]);

    // execution configuration
    dim3 block(blockSize);
    dim3 grid((nElems + block.x - 1) / block.x);

    // allocate the host memory
    float *h_A, *h_B;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);

    // initial data
    initialData(h_A, nElems);
    initialData(h_B, nElems);

    // allocate the device memory
    float *d_A, *d_B, *d_C;
    CHECK_ERROR(hipMalloc((void **)&d_A, nBytes));
    CHECK_ERROR(hipMalloc((void **)&d_B, nBytes));
    CHECK_ERROR(hipMalloc((void **)&d_C, nBytes));

    // transfer data from host to device
    CHECK_ERROR(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    char *token;
    int offset;
    token = strtok(input, ",");
    while (token != NULL)
    {
        offset = atoi(token);
        // launch the kernel 1 : Read Offset
        printf("Launch Kernel<<<%d,%d>>> with %d offset\n", grid.x, block.x, offset);
        readOffset<<<grid, block>>>(d_A, d_B, d_C, nElems, offset);
        CHECK_ERROR(hipDeviceSynchronize());

        // Launch Kernel 2 : Read only Cache
        printf("Launch Kernel<<<%d,%d>>> with %d offset\n", grid.x, block.x, offset);
        readOnlyCache<<<grid, block>>>(d_A, d_B, d_C, nElems, offset);
        CHECK_ERROR(hipDeviceSynchronize());

        token = strtok(NULL, ",");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);

    CHECK_ERROR(hipDeviceReset());
    return EXIT_SUCCESS;
}