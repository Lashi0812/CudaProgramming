#include "../include/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MEGABYTE    (1024 * 1024)

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CHECK_ERROR(hipSetDevice(dev));

    if (argc != 2) {
        printf("usage: %s <size-in-mbs>\n", argv[0]);
        return 1;
    }

    // memory size
    int n_mbs = atoi(argv[1]);
    unsigned int nbytes = n_mbs * MEGABYTE;

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK_ERROR(hipGetDeviceProperties(&deviceProp, dev));

    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK_ERROR(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("%s starting at ", argv[0]);
    printf("device %d: %s nbyte %5.2fMB canMap %d\n", dev,
           deviceProp.name, nbytes / (1024.0f * 1024.0f),
           deviceProp.canMapHostMemory);

    // allocate pinned host memory
    float *h_a;
    double start = seconds();
    CHECK_ERROR(hipHostMalloc ((float **)&h_a, nbytes));
    double elapsed = seconds() - start;
    printf("Host memory allocation took %2.10f us\n", elapsed * 1000000.0);

    // allocate device memory
    float *d_a;
    CHECK_ERROR(hipMalloc((float **)&d_a, nbytes));

    // initialize host memory
    memset(h_a, 0, nbytes);

    for (int i = 0; i < nbytes / sizeof(float); i++) h_a[i] = 100.10f;

    // transfer data from the host to the device
    CHECK_ERROR(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK_ERROR(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK_ERROR(hipFree(d_a));
    start = seconds();
    CHECK_ERROR(hipHostFree(h_a));
    elapsed = seconds() - start;
    printf("Host memory deallocation took %2.10f us\n", elapsed * 1000000.0);

    // reset device
    CHECK_ERROR(hipDeviceReset());
    return EXIT_SUCCESS;
}