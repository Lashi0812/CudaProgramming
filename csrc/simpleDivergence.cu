#include "hip/hip_runtime.h"
#include "../include/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// this to avoid the overhead of first run
__global__ void warmingUp(float *c)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if (tid % 2 == 0)
    {
        ia = 200.0f;
    }
    else
    {
        ib = 100.0f;
    }
    c[tid] = ia + ib;
}

__global__ void kernel1(float *c)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if (tid % 2 == 0)
    {
        ia = 200.0f;
    }
    else
    {
        ib = 100.0f;
    }
    c[tid] = ia + ib;
}

__global__ void kernel2(float *c)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 200.0f;
    }
    else
    {
        ib = 100.0f;
    }
    c[tid] = ia + ib;
}

__global__ void kernel3(float *c)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    bool pred = (tid % 2 == 0);
    if (pred)
    {
        ia = 200.f;
    }
    if(!pred)
    {
        ib = 100.0f;
    }
    c[tid] = ia + ib;
}

int main(int argc, char *argv[])
{
    // setup the device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s using device %d : %s\n", argv[0], dev, deviceProp.name);

    // set time measure
    // hipEvent_t start, stop;
    // float elaspedTime;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    // set up data size
    int size = atoi(argv[1]);
    printf("Data size : %d \n", size);

    // execution configuration
    dim3 block(atoi(argv[2]));
    dim3 grid((size + block.x - 1) / block.x);
    printf("Execution configuration <<<%d,%d>>> \n", grid.x, block.x);

    // allocate the gpu memory
    float *d_C;
    size_t nBytes = size * sizeof(float);
    hipMalloc((float **)&d_C, nBytes);
    hipMemset(d_C, 0, size);

    // host memory
    float *h_C;
    h_C = (float *)malloc(nBytes);
    memset(h_C, 0, size);

    // run the warmup kernel to remove the overhead
    // hipEventRecord(start,0);
    warmingUp<<<grid, block>>>(d_C);
    // hipEventRecord(stop,0);
    // CHECK(hipEventSynchronize(stop));

    // hipEventElapsedTime(&elaspedTime, start,stop);
    // printf("Kernel 2       <<<%d,%d>>> elapsed %f ms \n",
    //        grid.x, block.x, elaspedTime);
    // CHECK(hipGetLastError());

    // run  kernel 1
    // hipEventRecord(start,0);
    kernel1<<<grid, block>>>(d_C);
    // hipEventRecord(stop,0);
    // CHECK(hipEventSynchronize(stop));

    // hipEventElapsedTime(&elaspedTime, start,stop);
    // printf("Kernel 2       <<<%d,%d>>> elapsed %f ms \n",
    //        grid.x, block.x, elaspedTime);
    // CHECK(hipGetLastError());

    // run  kernel 2
    // hipEventRecord(start,0);
    kernel2<<<grid, block>>>(d_C);
    // hipEventRecord(stop,0);
    // CHECK(hipEventSynchronize(stop));

    // hipEventElapsedTime(&elaspedTime, start,stop);
    // printf("Kernel 2       <<<%d,%d>>> elapsed %f ms \n",
    //        grid.x, block.x, elaspedTime);
    // CHECK(hipGetLastError());

    kernel3<<<grid, block>>>(d_C);

    // Free Gpu memory
    CHECK(hipFree(d_C));
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
