#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <iostream>

__global__ void index(int *matrix, int height, int width)
{
    // find the matrix original co-ordinate (row,col)
    int mat_row = blockIdx.x * blockDim.x + threadIdx.x;
    int mat_col = blockIdx.y * blockDim.y + threadIdx.y;

    // convert the matrix co-ordinate into linear index
    int mat_idx = mat_row * width + mat_col;

    // find the thread linear index at each block level
    int blk_th_idx = threadIdx.x * blockDim.y + threadIdx.y;
    int blk_th_row = blk_th_idx / blockDim.y;
    int blk_th_col = blk_th_idx % blockDim.y;

    // find the thread linear index at each TRANSPOSED block level
    int blk_trans_th_idx = threadIdx.y * blockDim.x + threadIdx.x;
    int blk_trans_th_row = blk_trans_th_idx / blockDim.x;
    int blk_trans_th_col = blk_trans_th_idx % blockDim.x;
    if (mat_idx == 0)
        printf("Grid Dim (%d,%d) and block Dim (%d,%d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

    printf("(%d,%d) \t(%d,%d) \t(%d,%d,%d) \t(%d,%d,%d) \t(%d,%d,%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, mat_row, mat_col, mat_idx, blk_th_row, blk_th_col, blk_th_idx,
           blk_trans_th_row, blk_trans_th_col, blk_trans_th_idx);
}

int main()
{
    int width = 9, height = 4;
    int nElems = height * width;
    int nBytes = nElems * sizeof(int);
    at::Tensor matrix = at::arange(nElems, at::kInt).reshape({height, width});

    std::cout << matrix << std::endl;
    std::cout << matrix.transpose(1, 0) << std::endl;

    std::cout << matrix.flatten() << std::endl;
    std::cout << matrix.transpose(1, 0).flatten() << std::endl;

    // std::cout << matrix[1][2] << std::endl;

    // device
    int *d_mat;
    hipMalloc((int **)&d_mat, nBytes);

    hipMemcpy(d_mat, matrix.data_ptr(), nBytes, hipMemcpyHostToDevice);

    dim3 block(2, 3);
    dim3 grid((height + block.x - 1) / block.x, (width + block.y - 1) / block.y);

    index<<<grid, block>>>(d_mat, height, width);
    hipDeviceSynchronize();

    hipFree(d_mat);
    hipDeviceReset();
}